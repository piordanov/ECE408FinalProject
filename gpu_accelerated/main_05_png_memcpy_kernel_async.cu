/*  Project:                ECE 408 Final Project
 *  File Name:              main.cu
 *  Calls:                  pngrw.cpp
 *  Called by:              none
 *  Associated Header:      ece408_final_gpu.h
 *  Date created:           Wed Nov 18 2015
 *  Engineers:              Peter Iordanov, Laura Galbraith, Conor Gardner
 *  Compiler:               nvcc
 *  Target OS:              Ubuntu Linux 14.04
 *  Target architecture:    CPU: x86 (64 bit)       GPU: Nvida GTX 970 (Compute Capability 5.2) */

#include <cstdio>
#include <thread>
#include "ece408_final_gpu.h"
#include "pngrw.h"

#define PNG_THREADS 8

inline void swap_buffers(unsigned char** a, unsigned char** b)
{
    unsigned char* swap = *a;
    *a = *b;
    *b = swap;
}

int main(int argc, char** argv)
{
    
    /* Stage 1 - parse command line arguments */
    
    // abort if 3 arguments were not passed
    if (argc != 4)
    {
        fprintf
        (
            stderr,
            "Error, invalid command line arguments\n"
            "Usage: %s [inital_grid.png] [output_pattern] [iterations]",
            argv[0]
        );
        return -1;
    }
    
    // abort for an invalid iteration count
    unsigned iterations = atoi(argv[3]);
    if (iterations == 0)
    {
        fprintf(stderr, "Error, iteration count must be at least 1\n");
        return -1;
    }
    
    /* Stage 2 - load starting cell grid and copy it to the GPU */
    
    // Black pixels are dead (0).  Else a cell is live (1)
    unsigned char* send_grid_h = NULL; // starting off NULL tells pngrw to allocate memory for us
    unsigned width;     // the number of cells (not bytes) across a row of send_grid
    unsigned height;    // the number of cells down a column of inital_grid
    unsigned grid_bytes = pngrw_read_file(&send_grid_h, &width, &height, argv[1]);
    if (grid_bytes == 0)
        // failed to read png file.  pngrw will print an error message for us
        return -1;

    // allocate double-buffered GPU memory for input and ouput grids
    unsigned char* read_grid_d;
    unsigned char* write_grid_d;
    hipMalloc(&read_grid_d, grid_bytes * sizeof(char));
    hipMalloc(&write_grid_d, grid_bytes * sizeof(char));
    
    double start = get_timestamp();
    
    /* Stage 3 - Copy the initial grid to the GPU and simulate the first generation */
    
    dim3 threads_per_block(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    dim3 blocks_per_grid
    (
        (width - 1) / (THREADS_PER_BLOCK_X * 8) + 1,
        (height - 1) / THREADS_PER_BLOCK_Y + 1,
        1
    );
    
    hipMemcpy(read_grid_d, send_grid_h, grid_bytes * sizeof(char), hipMemcpyHostToDevice);
    kernel<<<blocks_per_grid, threads_per_block>>>(read_grid_d, write_grid_d, width, height);
    hipDeviceSynchronize();
    swap_buffers(&read_grid_d, &write_grid_d);
    
    // we're done with the CPU's copy of the input grid
    delete[] send_grid_h;
    
    /* Stage 4 - Simulate all further simulations on the GPU
     * Overlap CPU execution, device --> host memory copies, and png encodings */
    
    // allocate page-locked memory to recieve frames from the GPU
    unsigned char* recv_grid_h[PNG_THREADS];
    for (unsigned ix = 0; ix < PNG_THREADS; ix++)
        hipHostMalloc(&(recv_grid_h[ix]), grid_bytes * sizeof(char));
    
    // generates a name such as output_000.png on the heap
    // if argv[2] = "output" and 99 < iterations < 1000, for example
    output_filename_t output_filename(argv[2], iterations);
    unsigned filename_len = output_filename.get_len();
    char thread_filenames[PNG_THREADS][filename_len + 1];
    
    hipStream_t kernel_stream;
    hipStream_t memcpy_stream;
    hipStreamCreate(&kernel_stream);
    hipStreamCreate(&memcpy_stream);
    
    std::thread encoder[PNG_THREADS];
    
    for (unsigned gen_ix = 2; gen_ix < iterations; gen_ix++)
    {
        
        unsigned encoder_ix = gen_ix % PNG_THREADS;
        
        kernel
            <<<blocks_per_grid, threads_per_block, 0, kernel_stream>>>
            (read_grid_d, write_grid_d, width, height);
        
        if (encoder[encoder_ix].joinable())
            // if that CPU thread was already doing some work, wait for it to finish
            encoder[encoder_ix].join();

        // copy result from *previous* kernel launch in parallel with above kernel execution
        hipMemcpyAsync
        (
            recv_grid_h[encoder_ix],
            read_grid_d,
            grid_bytes * sizeof(char),
            hipMemcpyDeviceToHost,
            memcpy_stream
        );
        
        hipDeviceSynchronize();
        
        output_filename.next_filename();
        memcpy(&(thread_filenames[encoder_ix][0]), output_filename.str, filename_len + 1);
        
//        pngrw_write_file(thread_filenames[encoder_ix], recv_grid_h[encoder_ix], width, height, 1);
        
        // spawn a seperate CPU thread to do the PNG encoding
        encoder[encoder_ix] = std::thread
        (
            pngrw_write_file,
            &(thread_filenames[encoder_ix][0]), recv_grid_h[encoder_ix], width, height, 1
        );
        
        swap_buffers(&read_grid_d, &write_grid_d);
        
    }
    
    for (unsigned ix = 0; ix < PNG_THREADS; ix++)
        if (encoder[ix].joinable())
            encoder[ix].join();
    
    hipMemcpy
    (
        recv_grid_h[0],
        read_grid_d,
        grid_bytes * sizeof(char),
        hipMemcpyDeviceToHost
    );
    pngrw_write_file(output_filename.str, recv_grid_h[0], width, height);
    
    double elapsed_sec = get_timestamp() - start;
    
    /* Stage 5 - cleanup */
    
    // cleanup
    
    hipStreamDestroy(kernel_stream);
    hipStreamDestroy(memcpy_stream);
    
    for (unsigned ix = 0; ix < PNG_THREADS; ix++)
        hipHostFree(recv_grid_h[ix]);
    
    hipFree(read_grid_d);
    hipFree(write_grid_d);
    
    printf
    (
        "Success! Finished %s ~ %u iterations in %lf seconds ~ %lf cells/sec\n",
        argv[1],
        iterations,
        elapsed_sec,
        ((double)width * (double)height * (double)iterations) / elapsed_sec
    );
    
    return 0;
    
}

