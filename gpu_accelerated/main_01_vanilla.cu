/*  Project:                ECE 408 Final Project
 *  File Name:              main.cu
 *  Calls:                  pngrw.cpp
 *  Called by:              none
 *  Associated Header:      ece408_final_gpu.h
 *  Date created:           Tues Nov 10 2015
 *  Engineers:              Peter Iordanov, Laura Galbraith, Conor Gardner
 *  Compiler:               nvcc
 *  Target OS:              Ubuntu Linux 14.04
 *  Target architecture:    CPU: x86 (64 bit)       GPU: Nvida GTX 970 */

#include <cstdio>
#include "ece408_final_gpu.h"
#include "pngrw.h"

int main(int argc, char** argv)
{
    
    /* Stage 1 - parse command line arguments */
    
    // abort if 3 arguments were not passed
    if (argc != 4)
    {
        fprintf
        (
            stderr,
            "Error, invalid command line arguments\n"
            "Usage: %s [inital_grid.png] [output_pattern] [iterations]",
            argv[0]
        );
        return -1;
    }
    
    // abort for an invalid iteration count
    unsigned iterations = atoi(argv[3]);
    if (iterations == 0)
    {
        fprintf(stderr, "Error, iteration count must be at least 1\n");
        return -1;
    }
    
    /* Stage 2 - load starting cell grid and copy it to the GPU */
    
    // Black pixels are dead (0).  Else a cell is live (1)
    unsigned char* send_grid_h = NULL; // starting off NULL tells pngrw to allocate memory for us
    unsigned width;     // the number of cells (not bytes) across a row of send_grid
    unsigned height;    // the number of cells down a column of inital_grid
    unsigned grid_bytes = pngrw_read_file(&send_grid_h, &width, &height, argv[1]);
    if (grid_bytes == 0)
        // failed to read png file.  pngrw will print an error message for us
        return -1;

    // allocate double-buffered GPU memory for input and ouput grids
    unsigned char* read_grid_d;
    unsigned char* write_grid_d;
    hipMalloc(&read_grid_d, grid_bytes * sizeof(char));
    hipMalloc(&write_grid_d, grid_bytes * sizeof(char));
    
    double start = get_timestamp();
    
    // blocking copy initial grid to GPU
    hipMemcpy(read_grid_d, send_grid_h, grid_bytes * sizeof(char), hipMemcpyHostToDevice);

    // we're done with the CPU's copy of the input grid
    delete[] send_grid_h;
    
    /* Stage 3 - Simulate each generation of cells on the GPU */
    
    // allocate page-locked memory to recieve frames from the GPU
    unsigned char* recv_grid_h;
    hipHostMalloc(&recv_grid_h, grid_bytes * sizeof(char));
    
    // generates a name such as output_000.png on the heap
    // if argv[2] = "output" and 99 < iterations < 1000, for example
    output_filename_t output_filename(argv[2], iterations);
    
    // use the GPU to simulate all iterations in the most inefficient way possible (no overlap)
    dim3 threads_per_block(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1);
    dim3 blocks_per_grid
    (
        (width / 8 - 1) / THREADS_PER_BLOCK_X + 1,
        (height - 1) / THREADS_PER_BLOCK_Y + 1,
        1
    );
    
    for (unsigned gen_ix = 0; gen_ix < iterations; gen_ix++)
    {
        
        output_filename.next_filename();
        kernel<<<blocks_per_grid, threads_per_block>>>(read_grid_d, write_grid_d, width, height);
        hipMemcpy(recv_grid_h, write_grid_d, grid_bytes * sizeof(char), hipMemcpyDeviceToHost);
        pngrw_write_file(output_filename.str, recv_grid_h, width, height);
        
        unsigned char* swap = read_grid_d;
        read_grid_d = write_grid_d;
        write_grid_d = swap;

    }
    
    double elapsed_sec = get_timestamp() - start;
    
    /* Stage 4 - cleanup */
    
    // cleanup
    hipHostFree(recv_grid_h);
    hipFree(read_grid_d);
    hipFree(write_grid_d);
    
    printf
    (
        "Success! Finished %s ~ %u iterations in %lf seconds ~ %lf cells/sec\n",
        argv[1],
        iterations,
        elapsed_sec,
        ((double)width * (double)height * (double)iterations) / elapsed_sec
    );
    
    return 0;
    
}

